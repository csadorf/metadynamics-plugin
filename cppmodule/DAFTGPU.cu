#include "hip/hip_runtime.h"
#include "DAFTGPU.cuh"

__device__ uint3 get_grid_pos(unsigned int grid_idx,
                              unsigned int nx,
                              unsigned int ny,
                              unsigned int nz)
    {
    uint3 pos;
    pos.x = grid_idx/ny/nz;
    pos.y = (grid_idx - pos.x*ny*nz)/nz;
    pos.z = (grid_idx - pos.x*ny*nz - pos.y*nz);
    return pos;
    }

__device__ unsigned int get_grid_idx(uint3 grid_pos,
                              unsigned int nx,
                              unsigned int ny,
                              unsigned int nz)
    {
    unsigned int idx = grid_pos.z + nz * (grid_pos.y + ny * grid_pos.x);
    return idx;
    }

__global__ void gpu_combine_buf_kernel(unsigned int n_cells,
                          hipfftComplex *d_combine_buf,
                          const hipfftComplex *d_stage_buf,
                          const bool sw,
                          const unsigned int n_current_dir,
                          const unsigned int stride)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n_cells) return;

    if (sw)
        {
        hipfftComplex local = d_combine_buf[idx];
        hipfftComplex remote = d_stage_buf[idx];
        local.x += remote.x;
        local.y += remote.y;
        d_combine_buf[idx] = local;
        }
    else
        {
        hipfftComplex local = d_combine_buf[idx];
        local.x *= Scalar(-1.0);
        local.y *= Scalar(-1.0);
        hipfftComplex remote = d_stage_buf[idx];
        hipfftComplex exp_fac, out;
        unsigned int grid_idx = idx % stride;
        exp_fac.x = cosf(Scalar(2.0*M_PI)*(Scalar)grid_idx/(Scalar)n_current_dir);
        exp_fac.y = sinf(Scalar(2.0*M_PI)*(Scalar)grid_idx/(Scalar)n_current_dir);
        out.x = exp_fac.x * (local.x + remote.x) - exp_fac.y * (local.y + remote.y);
        out.y = exp_fac.x * (local.y + remote.y) + exp_fac.y * (local.x + remote.x);
        d_combine_buf[idx] = out;
        } 
    }

void gpu_combine_buf(unsigned int n_cells,
                     hipfftComplex *d_combine_buf,
                     const hipfftComplex *d_stage_buf,
                     const bool sw,
                     const unsigned int n_current_dir,
                     const unsigned int stride)
    {
    unsigned int block_size = 512;
    unsigned int n_blocks = n_cells/block_size;
    if (n_cells % block_size) n_blocks+=1;
    gpu_combine_buf_kernel<<<n_blocks, block_size>>>(n_cells,  d_combine_buf, d_stage_buf, sw, n_current_dir, stride);
    }

__global__ void gpu_rotate_buf_kernel_z_y(unsigned int nx,
                                      unsigned int ny,
                                      unsigned int nz,
                                      const hipfftComplex *d_combine_buf,
                                      hipfftComplex *d_work_buf)
    {
    unsigned int idx = blockIdx.x * blockDim. x + threadIdx.x;

    if (idx >= nx*ny*nz) return;

    uint3 grid_pos = get_grid_pos(idx, nx, ny, nz);
    d_work_buf[grid_pos.y + ny * (grid_pos.x + nx * grid_pos.z)] =
        d_combine_buf[grid_pos.z + nz * (grid_pos.y + ny * grid_pos.x)];
    }

__global__ void gpu_rotate_buf_kernel_y_x(unsigned int nx,
                                      unsigned int ny,
                                      unsigned int nz,
                                      const hipfftComplex *d_combine_buf,
                                      hipfftComplex *d_work_buf)
    {
    unsigned int idx = blockIdx.x * blockDim. x + threadIdx.x;

    if (idx >= nx*ny*nz) return;

    uint3 grid_pos = get_grid_pos(idx, nx, ny, nz);
    d_work_buf[grid_pos.x + nx * (grid_pos.z + nz * grid_pos.y)] =
        d_combine_buf[grid_pos.y + ny * (grid_pos.x + nx * grid_pos.z)];
    } 

void gpu_rotate_buf_z_y(unsigned int nx,
                        unsigned int ny,
                        unsigned int nz,
                        const hipfftComplex *d_combine_buf,
                        hipfftComplex *d_work_buf)
    {
    unsigned int block_size = 512;
    unsigned int n_cells = nx*ny*nz;
    unsigned int n_blocks = n_cells/block_size;
    if (n_cells % block_size) n_blocks+=1;

    gpu_rotate_buf_kernel_z_y<<<n_blocks, block_size>>>(nx, ny, nz, d_combine_buf, d_work_buf);
    }

void gpu_rotate_buf_y_x(unsigned int nx,
                        unsigned int ny,
                        unsigned int nz,
                        const hipfftComplex *d_combine_buf,
                        hipfftComplex *d_work_buf)
    {
    unsigned int block_size = 512;
    unsigned int n_cells = nx*ny*nz;
    unsigned int n_blocks = n_cells/block_size;
    if (n_cells % block_size) n_blocks+=1;

    gpu_rotate_buf_kernel_y_x<<<n_blocks, block_size>>>(nx, ny, nz, d_combine_buf, d_work_buf);
    } 
