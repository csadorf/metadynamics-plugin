#include "hip/hip_runtime.h"
/*! \file LamellarOrderParameterGPU.cu
    \brief CUDA implementation of LamellarOrderParameter GPU routines
 */
#include <hip/hip_runtime.h>

#include "LamellarOrderParameterGPU.cuh"

__global__ void kernel_calculate_sq_partial(
            int n_particles,
            Scalar *fourier_mode_partial,
            Scalar4 *postype,
            int n_wave,
            Scalar3 *wave_vectors,
            Scalar *d_modes,
            Scalar *phases)
    {
    extern __shared__ Scalar sdata[];

    unsigned int tidx = threadIdx.x;

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned int i = 0; i < n_wave; i++) {
        Scalar3 q = wave_vectors[i];
        Scalar phi = phases[i];

        Scalar mySum = Scalar(0.0);

        if (j < n_particles) {

            Scalar3 p = make_scalar3(postype[j].x, postype[j].y, postype[j].z);
            Scalar dotproduct = q.x * p.x + q.y * p.y + q.z * p.z;
            unsigned int type = __float_as_int(postype[j].w);
            Scalar mode = d_modes[type];
            mySum +=mode*cosf(dotproduct+phi);
        }
        sdata[tidx] = mySum;

       __syncthreads();
        // reduce in shared memory
        if (blockDim.x >= 512)
            {
            if (tidx < 256)
                {
                mySum += sdata[tidx+256];
                sdata[tidx] = mySum;
                }
            __syncthreads();
            }

        if (blockDim.x >= 256) {
            if (tidx < 128)
                {
                mySum += sdata[tidx+128];
                sdata[tidx] = mySum;
                }
            __syncthreads();
            }

        if (blockDim.x >= 128)
            {
            if (tidx < 64)
                {
                mySum += sdata[tidx+64];
                sdata[tidx] = mySum;
                }
           __syncthreads();
            }

        if (tidx < 32) {
            volatile Scalar* smem = sdata;
            if (blockDim.x >= 64)
                {
                Scalar rhs;
                rhs = mySum + smem[tidx + 32];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            if (blockDim.x >= 32)
                {
                Scalar rhs;
                rhs = mySum + smem[tidx + 16];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            if (blockDim.x >= 16)
                {
                Scalar rhs;
                rhs = mySum + smem[tidx + 8];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            if (blockDim.x >=  8)
                {
                Scalar rhs;
                rhs = mySum + smem[tidx + 4];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            if (blockDim.x >=  4)
                {
                Scalar rhs;
                rhs = mySum + smem[tidx + 2];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            if (blockDim.x >=  2)
                { 
                Scalar rhs;
                rhs = mySum + smem[tidx + 1];
                smem[tidx] = rhs;
                mySum = rhs;
                }
            }

        // write result to global memeory
        if (tidx == 0)
           fourier_mode_partial[blockIdx.x + gridDim.x*i] = sdata[0];
        } // end loop over wave vectors
    } 

__global__ void kernel_final_reduce_fourier_modes(Scalar* fourier_mode_partial,
                                       unsigned int nblocks,
                                       Scalar *fourier_modes,
                                       unsigned int n_wave)
    {
    extern __shared__ volatile Scalar smem[];

    for (unsigned int j = 0; j < n_wave; ++j)
        {
        if (threadIdx.x == 0)
           fourier_modes[j] = Scalar(0.0);

        for (int start = 0; start< nblocks; start += blockDim.x)
            {
            __syncthreads();
            if (start + threadIdx.x < nblocks)
                smem[threadIdx.x] = fourier_mode_partial[j*nblocks+start + threadIdx.x];
            else
                smem[threadIdx.x] = Scalar(0.0);

            __syncthreads();

            // reduce the sum
            int offs = blockDim.x >> 1;
            while (offs > 0)
                {
                if (threadIdx.x < offs)
                    smem[threadIdx.x] += smem[threadIdx.x + offs];
                offs >>= 1;
                __syncthreads();
                }

             if (threadIdx.x == 0)
                fourier_modes[j] += smem[0];
            }
        }
    }

hipError_t gpu_calculate_fourier_modes(unsigned int n_wave,
                                 Scalar3 *d_wave_vectors,
                                 unsigned int n_particles,
                                 Scalar4 *d_postype,
                                 Scalar *d_mode,
                                 Scalar *d_fourier_modes,
                                 Scalar *d_phases,
                                 unsigned int block_size,
                                 Scalar *d_fourier_mode_partial
                                 )
    {
    hipError_t cudaStatus;

    unsigned int n_blocks = n_particles/block_size + 1;

    unsigned int shared_size = block_size * sizeof(Scalar);
    kernel_calculate_sq_partial<<<n_blocks, block_size, shared_size>>>(
               n_particles,
               d_fourier_mode_partial,
               d_postype,
               n_wave,
               d_wave_vectors,
               d_mode,
               d_phases);

    if (cudaStatus = hipGetLastError()) 
           return cudaStatus;

    // calculate final S(q) values 
    const unsigned int final_block_size = 512;
    shared_size = final_block_size*sizeof(Scalar);
    kernel_final_reduce_fourier_modes<<<1, final_block_size,shared_size>>>(d_fourier_mode_partial,
                                                                  n_blocks,
                                                                  d_fourier_modes,
                                                                  n_wave);
                                                                  

    if (cudaStatus = hipGetLastError())
        return cudaStatus;

    return hipSuccess;
    }

__global__ void kernel_compute_sq_forces(unsigned int N,
                                  Scalar4 *postype,
                                  Scalar4 *force,
                                  unsigned int n_wave,
                                  Scalar3 *wave_vectors,
                                  Scalar *mode,
                                  Scalar n_global,
                                  Scalar bias,
                                  Scalar *phases)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    Scalar4 p = postype[idx];
    Scalar3 pos = make_scalar3(p.x, p.y, p.z);
    unsigned int type = __float_as_int(p.w);

    Scalar m = mode[type];

    Scalar4 force_energy = make_scalar4(0.0f,0.0f,0.0f,0.0f);

    for (unsigned int k = 0; k < n_wave; k++)
        {
        Scalar3 q = wave_vectors[k];
        Scalar dotproduct = dot(pos,q);

        Scalar f = m*sinf(dotproduct + phases[k]);
        
        force_energy.x += q.x*f;
        force_energy.y += q.y*f;
        force_energy.z += q.z*f;
        }

    force_energy.x /= n_global;
    force_energy.y /= n_global;
    force_energy.z /= n_global;

    force_energy.x *= bias;
    force_energy.y *= bias;
    force_energy.z *= bias;

    force[idx] = force_energy;
    }

hipError_t gpu_compute_sq_forces(unsigned int N,
                                  Scalar4 *d_postype,
                                  Scalar4 *d_force,
                                  unsigned int n_wave,
                                  Scalar3 *d_wave_vectors,
                                  Scalar *d_mode,
                                  unsigned int n_global,
                                  Scalar bias,
                                  Scalar *d_phases)
    {
    hipError_t cudaStatus;
    const unsigned int block_size = 512;

    kernel_compute_sq_forces<<<N/block_size + 1, block_size>>>(N,
                                                               d_postype,
                                                               d_force,
                                                               n_wave,
                                                               d_wave_vectors,
                                                               d_mode,
                                                               n_global,
                                                               bias,
                                                               d_phases);

    cudaStatus = hipGetLastError();
    return cudaStatus;
    }
