#include "hip/hip_runtime.h"
/*! \file LamellarOrderParameterGPU.cu
    \brief CUDA implementation of LamellarOrderParameter GPU routines
 */
#include <hip/hip_runtime.h>

#include "LamellarOrderParameterGPU.cuh"

__global__ void kernel_calculate_sq_partial(
            int n_particles,
            Scalar2 *fourier_mode_partial,
            Scalar4 *postype,
            int n_wave,
            Scalar3 *wave_vectors,
            Scalar *d_modes,
            Scalar *phases)
    {
    extern __shared__ Scalar2 sdata[];

    unsigned int tidx = threadIdx.x;

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned int i = 0; i < n_wave; i++) {
        Scalar3 q = wave_vectors[i];
        Scalar phi = phases[i];

        Scalar2 mySum = make_scalar2(0.0f,0.0f);

        if (j < n_particles) {

            Scalar3 p = make_scalar3(postype[j].x, postype[j].y, postype[j].z);
            Scalar dotproduct = q.x * p.x + q.y * p.y + q.z * p.z;
            unsigned int type = __float_as_int(postype[j].w);
            Scalar mode = d_modes[type];
            Scalar2 exponential = make_scalar2(mode*cosf(dotproduct+phi),
                                                   mode*sinf(dotproduct+phi));
            mySum.x += exponential.x;
            mySum.y += exponential.y;
        }
        sdata[tidx] = mySum;

       __syncthreads();
        // reduce in shared memory
        if (blockDim.x >= 512)
            {
            if (tidx < 256)
                {
                mySum.x += sdata[tidx+256].x;
                mySum.y += sdata[tidx+256].y;
                sdata[tidx] = mySum;
                }
            __syncthreads();
            }

        if (blockDim.x >= 256) {
            if (tidx < 128)
                {
                mySum.x += sdata[tidx+128].x;
                mySum.y += sdata[tidx+128].y;
                sdata[tidx] = mySum;
                }
            __syncthreads();
            }

        if (blockDim.x >= 128)
            {
            if (tidx < 64)
                {
                mySum.x += sdata[tidx+64].x;
                mySum.y += sdata[tidx+64].y;
                sdata[tidx] = mySum;
                }
           __syncthreads();
            }

        if (tidx < 32) {
            volatile Scalar2* smem = sdata;
            if (blockDim.x >= 64)
                {
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 32].x;
                rhs.y = mySum.y + smem[tidx + 32].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            if (blockDim.x >= 32)
                {
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 16].x;
                rhs.y = mySum.y + smem[tidx + 16].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            if (blockDim.x >= 16)
                {
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 8].x;
                rhs.y = mySum.y + smem[tidx + 8].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            if (blockDim.x >=  8)
                {
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 4].x;
                rhs.y = mySum.y + smem[tidx + 4].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            if (blockDim.x >=  4)
                {
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 2].x;
                rhs.y = mySum.y + smem[tidx + 2].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            if (blockDim.x >=  2)
                { 
                Scalar2 rhs;
                rhs.x = mySum.x + smem[tidx + 1].x;
                rhs.y = mySum.y + smem[tidx + 1].y;
                smem[tidx].x = rhs.x;
                smem[tidx].y = rhs.y;
                mySum = rhs;
                }
            }

        // write result to global memeory
        if (tidx == 0)
           fourier_mode_partial[blockIdx.x + gridDim.x*i] = sdata[0];
        } // end loop over wave vectors
    } 

__global__ void kernel_final_reduce_fourier_modes(Scalar2* fourier_mode_partial,
                                       unsigned int nblocks,
                                       Scalar2 *fourier_modes,
                                       unsigned int n_wave)
    {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_wave)
        return;

    // do final reduction of fourier mode
    Scalar2 fourier_mode = make_scalar2(0.0f,0.0f);
    for (unsigned int j = 0; j < nblocks; j++)
        { 
        fourier_mode.x += fourier_mode_partial[j + i*nblocks].x;
        fourier_mode.y += fourier_mode_partial[j + i*nblocks].y;
        }

    fourier_modes[i] = make_scalar2(fourier_mode.x, fourier_mode.y); 
    }

hipError_t gpu_calculate_fourier_modes(unsigned int n_wave,
                                 Scalar3 *d_wave_vectors,
                                 unsigned int n_particles,
                                 Scalar4 *d_postype,
                                 Scalar *d_mode,
                                 Scalar2 *d_fourier_modes,
                                 Scalar *d_phases
                                 )
    {
    Scalar2 *d_fourier_mode_partial;

    hipError_t cudaStatus;

    const unsigned int block_size_x = 256;
    unsigned int n_blocks_x = n_particles/block_size_x + 1;

    hipMalloc(&d_fourier_mode_partial, sizeof(Scalar2)*n_wave*n_blocks_x);

    unsigned int shared_size = block_size_x * sizeof(Scalar2);
    kernel_calculate_sq_partial<<<n_blocks_x, block_size_x, shared_size>>>(
               n_particles,
               d_fourier_mode_partial,
               d_postype,
               n_wave,
               d_wave_vectors,
               d_mode,
               d_phases);

    if (cudaStatus = hipGetLastError()) 
           return cudaStatus;

    // calculate final S(q) values 
    const unsigned int block_size = 512;
    kernel_final_reduce_fourier_modes<<<n_wave/block_size + 1, block_size>>>(d_fourier_mode_partial,
                                                                  n_blocks_x,
                                                                  d_fourier_modes,
                                                                  n_wave);
                                                                  

    if (cudaStatus = hipGetLastError())
        return cudaStatus;

    hipFree(d_fourier_mode_partial);

    return hipSuccess;
    }

__global__ void kernel_compute_sq_forces(unsigned int N,
                                  Scalar4 *postype,
                                  Scalar4 *force,
                                  unsigned int n_wave,
                                  Scalar3 *wave_vectors,
                                  Scalar *mode,
                                  Scalar n_global,
                                  Scalar bias,
                                  Scalar *phases)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    Scalar4 p = postype[idx];
    Scalar3 pos = make_scalar3(p.x, p.y, p.z);
    unsigned int type = __float_as_int(p.w);

    Scalar m = mode[type];

    Scalar4 force_energy = make_scalar4(0.0f,0.0f,0.0f,0.0f);

    for (unsigned int k = 0; k < n_wave; k++)
        {
        Scalar3 q = wave_vectors[k];
        Scalar dotproduct = dot(pos,q);

        Scalar f = m*sinf(dotproduct + phases[k]);
        
        force_energy.x += q.x*f;
        force_energy.y += q.y*f;
        force_energy.z += q.z*f;
        }

    force_energy.x /= n_global;
    force_energy.y /= n_global;
    force_energy.z /= n_global;

    force_energy.x *= bias;
    force_energy.y *= bias;
    force_energy.z *= bias;

    force[idx] = force_energy;
    }

hipError_t gpu_compute_sq_forces(unsigned int N,
                                  Scalar4 *d_postype,
                                  Scalar4 *d_force,
                                  unsigned int n_wave,
                                  Scalar3 *d_wave_vectors,
                                  Scalar *d_mode,
                                  unsigned int n_global,
                                  Scalar bias,
                                  Scalar *d_phases)
    {
    hipError_t cudaStatus;
    const unsigned int block_size = 512;

    kernel_compute_sq_forces<<<N/block_size + 1, block_size>>>(N,
                                                               d_postype,
                                                               d_force,
                                                               n_wave,
                                                               d_wave_vectors,
                                                               d_mode,
                                                               n_global,
                                                               bias,
                                                               d_phases);

    cudaStatus = hipGetLastError();
    return cudaStatus;
    }
